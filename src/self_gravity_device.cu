#include "hip/hip_runtime.h"
#define __GPU
#define __NOPROTO

#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>

#include "fargo3d.h"
 
typedef struct SG_data_gpu {  
#ifdef SELF_GRAVITY
  // These will exist on only one the CPU or GPU
  real* source;
  real* kernel;
  real* pot;

#ifdef GPU
  hipfftHandle fwd_cuplan ;
  hipfftHandle bwd_cuplan ;
#endif
  
  real soft ; // Softening parameter (softening length / R)
  real norm ; // Normalization for fft work
  real delta_u; // u-space separation between radial grid cells
  real delta_t; // phi-space separation between radial grid cells
  real Rmid0;   // Middle of first radial grid cell
#endif
} SGData ;
static SGData sg_data_gpu ;
static FFTGrid fft_grid ;

#ifdef GPU

#define ymin(i) ymin_s[(i)]
CONSTANT(real, ymin_s, 7686);

#ifdef FLOAT
typedef hipfftComplex complex ;
#else
typedef hipfftDoubleComplex complex ;
#endif

#endif

extern "C" void prs_exit(int) ;
extern "C" void CheckNans (char *string) ;

// Prototypes for internal functions
void sgkernel_gpu();
void sgsource_gpu() ;
void sgpot_gpu() ;

// Compute the potential due to self-gravity
extern "C" void SelfGravityPotential_gpu() {
#ifdef GPU
  sgsource_gpu() ;
  sgpot_gpu() ; 
#endif
}


extern "C" void SelfGravityInit_gpu() {
#ifdef GPU
#if defined(SELF_GRAVITY)
  /* Self-gravity only works in 2D cylindrical co-ordinates for now */

#ifndef CYLINDRICAL
#error Self-gravity requires cylindrical co-ordinates
#endif

#if !(defined (X) && defined(Y))
#error X and Y must be active for self-gravity
#endif

#ifdef Z
#error Self-gravity does not work with Z dimensions
#endif
  
  if (!((toupper(*SPACING) == 'L') && (toupper(*(SPACING+1)) == 'O'))) {
    fprintf(stderr, "Error: self-gravity requires a logarithmic grid\n");
    prs_exit(1) ;
  } 

#ifdef FLOAT
  if (hipfftPlan2d(&(sg_data_gpu.fwd_cuplan), 2*NY, NX, HIPFFT_R2C) != HIPFFT_SUCCESS) 
#else
  if (hipfftPlan2d(&(sg_data_gpu.fwd_cuplan), 2*NY, NX, HIPFFT_D2Z) != HIPFFT_SUCCESS) 
#endif
    {

      fprintf(stderr, "Error: failed to create fwd cufft_cuplan for SG") ;
      prs_exit(1) ;
    }

#ifdef FLOAT
  if (hipfftPlan2d(&(sg_data_gpu.bwd_cuplan), 2*NY, NX, HIPFFT_C2R) != HIPFFT_SUCCESS) 
#else
  if (hipfftPlan2d(&(sg_data_gpu.bwd_cuplan), 2*NY, NX, HIPFFT_Z2D) != HIPFFT_SUCCESS) 
#endif
    {
      fprintf(stderr, "Error: failed to create bwd cufft_cuplan for SG") ;
      prs_exit(1) ;
    }


  // Make sure that we have enough space for boundaries  
  size_t local_size ;
  hipfftGetSize(sg_data_gpu.fwd_cuplan, &local_size) ;

  hipMalloc((void**)&(sg_data_gpu.source), local_size) ;
  check_errors("SG_alloc");
  hipMalloc((void**)&(sg_data_gpu.kernel), local_size) ;
  check_errors("SG_alloc");
  hipMalloc((void**)&(sg_data_gpu.pot), local_size) ;
  check_errors("SG_alloc");

			   
  // Store grid sizes
  fft_grid.Ny = NX ;
  fft_grid.stride_y = (NX/2 + 1);
  fft_grid.local_Nx = 2*NY ;

  // Save the additional constants
  sg_data_gpu.soft = SELFGRAVITYSOFTENING;
  sg_data_gpu.delta_u = log(YMAX/YMIN) / NY ;
  sg_data_gpu.delta_t = 2*M_PI/NX ;
  sg_data_gpu.Rmid0 = (YMIN + YMIN*exp(sg_data_gpu.delta_u))/2 ;
  sg_data_gpu.norm =
    sg_data_gpu.delta_u*sg_data_gpu.delta_t * sg_data_gpu.Rmid0/(2*NX*NY);
  
  // Initialize the kernel
  sgkernel_gpu() ;
  printf("Initialised self-gravity\n") ;
#endif
#endif
}

///============================================================================
// Internal functions

#ifdef GPU

__global__  void __sgkernel(SGData sg_data, FFTGrid fft_grid)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < fft_grid.local_Nx && j < fft_grid.Ny) {
    real du   = sg_data.delta_u ;
    real dphi = sg_data.delta_t ;

    real eps = sg_data.soft*sg_data.soft ;

    int stride_fft = fft_grid.Ny ;
  
    // u runs over range [-umax, umax), where umax = log(YMAX/YMIN).
    real u = du * i ;
    if (i >= fft_grid.local_Nx/2) u -= fft_grid.local_Nx * du ;
      
    real phi = dphi * j ;
      
     sg_data.source[i*stride_fft + j] = 
       - G * sg_data.norm / sqrt(2*(cosh(u) - cos(phi)) + eps*exp(u)) ;
  }
}


// Setup the gravity kernel
void sgkernel_gpu() {
#ifdef SELF_GRAVITY
  dim3 block (BLOCK_X, BLOCK_Y);
  dim3 grid ((fft_grid.local_Nx+block.x-1)/block.x,
	     (fft_grid.Ny+block.y-1)/block.y) ;

  __sgkernel<<<grid,block>>>(sg_data_gpu, fft_grid) ;
  check_errors("__sgkernel") ;
      
  // Do the FFT of the kernel
#ifdef FLOAT
  hipfftExecR2C(sg_data_gpu.fwd_cuplan, sg_data_gpu.source, (complex*) sg_data_gpu.kernel);
#else
  hipfftExecD2Z(sg_data_gpu.fwd_cuplan, sg_data_gpu.source, (complex*) sg_data_gpu.kernel);
#endif
  check_errors("sgkernel_fft") ;
#endif
}

__global__ void __sgsource(SGData sg_data, FFTGrid fft_grid, 
			   real* rho, int pitch, int stride) {

  int ii = threadIdx.x + blockIdx.x * blockDim.x;
  int jj = threadIdx.y + blockIdx.y * blockDim.y;
  
  int stride_fft = fft_grid.Ny ;
  int Nx = fft_grid.local_Nx / 2 ; // Must be NY
  double du = sg_data.delta_u ;

  if (jj < fft_grid.Ny) {
    if (ii < Nx) {
      int i = 0, j=0, k = 0 ;
      i = jj + NGHX ; // Transposed
      j = ii + NGHY ;
    
      sg_data.pot[ii*stride_fft + jj] = rho[l] * exp(1.5 * du * ii) ;	
    }
    else {
      sg_data.pot[ii*stride_fft + jj] = 0 ;
    }
  }
}

// Compute the fft of the source term (density)
void sgsource_gpu() {
#ifdef SELF_GRAVITY
  INPUT(Density);

  dim3 block (BLOCK_X, BLOCK_Y);
  dim3 grid ((fft_grid.local_Nx+block.x-1)/block.x,
	     (fft_grid.Ny+block.y-1)/block.y) ;

  __sgsource<<<grid,block>>>(sg_data_gpu, fft_grid, Density->field_gpu,	
			     Pitch_gpu, Stride_gpu) ;
  check_errors("__sgsource") ;
      
  // Do the FFT of the source term
#ifdef FLOAT
  hipfftExecR2C(sg_data_gpu.fwd_cuplan, sg_data_gpu.pot, (complex*) sg_data_gpu.source) ;
#else
  hipfftExecD2Z(sg_data_gpu.fwd_cuplan, sg_data_gpu.pot, (complex*) sg_data_gpu.source) ;
#endif
  check_errors("sgsource_fft") ;
#endif
}

__global__ void __sgpot1(SGData sg_data, FFTGrid fft_grid) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  
  complex* src  = (complex*) sg_data.source ;
  complex* kern = (complex*) sg_data.kernel ;

  if (i < fft_grid.local_Nx && j < fft_grid.stride_y) {
    int idx = i*fft_grid.stride_y + j ;
    complex result ;
    result.x = src[idx].x*kern[idx].x - src[idx].y*kern[idx].y ;
    result.y = src[idx].x*kern[idx].y + src[idx].y*kern[idx].x ;
    //src[idx] = kern[idx] ;
    src[idx] =  result ;
  }

}

__global__ void __sgpot2(SGData sg_data, FFTGrid fft_grid,
			 real* pot, int size_x, int size_y,
			 int pitch, int stride) {
  
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = 0 ;

  if (i < size_x && j < size_y) {

    int ii = i - NGHX ;
    int jj = j - NGHY ;
    if (jj < 0) jj += fft_grid.local_Nx ;

#ifdef GHOSTSX
    if (ii <  0) ii += fft_grid.Ny ;
    if (ii > fft_grid.Ny) ii -= fft_grid.Ny ;
#endif

    int stride_fft = fft_grid.Ny ;
    real Rmid0 = sg_data.Rmid0 ;
    real sqrt_r = sqrt(sg_data.Rmid0/ymed(j)) ;

    pot[l] += sg_data.pot[jj*stride_fft + ii] * sqrt_r ;
  }
}

// Compute the potential via the convolution of source and kernel
void sgpot_gpu() {
#ifdef SELF_GRAVITY
  INPUT(Pot);
  OUTPUT(Pot);

#ifdef BIGMEM
#define ymin_d &Ymin_d
#endif
  CUDAMEMCPY(ymin_s, ymin_d, sizeof(real)*(Ny+2*NGHY+1), 0, \
	     hipMemcpyDeviceToDevice);

  // Multiply source by kernel
  dim3 block (BLOCK_X, BLOCK_Y);
  dim3 grid ((fft_grid.local_Nx+block.x-1)/block.x,
	     (fft_grid.stride_y+block.y-1)/block.y) ;

  __sgpot1<<<grid,block>>>(sg_data_gpu, fft_grid) ;
  check_errors("__sgpot1") ;

  // Do the FFT of the source term
#ifdef FLOAT
  hipfftExecC2R(sg_data_gpu.bwd_cuplan, (complex*) sg_data_gpu.source, sg_data_gpu.pot) ;
#else
  hipfftExecZ2D(sg_data_gpu.bwd_cuplan, (complex*) sg_data_gpu.source, sg_data_gpu.pot) ;
#endif
  check_errors("sgpot_fft") ;

  dim3 grid2 ((Nx+2*NGHX+block.x-1)/block.x,
	      ((Ny+2*NGHY)+block.y-1)/block.y) ;

  __sgpot2<<<grid2,block>>>(sg_data_gpu, fft_grid, Pot->field_gpu, 
			    Nx + 2*NGHX, Ny+2*NGHY,
			    Pitch_gpu, Stride_gpu) ;
  check_errors("__sgpot2") ;
#endif
}


#endif // GPU
